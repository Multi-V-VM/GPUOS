#include "hip/hip_runtime.h"
#include "common.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Global jump table and counters
__device__ __managed__ OpFn g_op_table[GPUOS_MAX_OPS];
__device__ __managed__ int  g_op_alias[GPUOS_MAX_OPS];
__device__ __managed__ unsigned long long g_processed_count = 0ULL;

// Built-in default operator: C = A + B
extern "C" __device__ void op_add(const Task& t) {
  const float* a = static_cast<const float*>(t.in0);
  const float* b = static_cast<const float*>(t.in1);
  float* c = static_cast<float*>(t.out0);
  int n = t.n;
  // Block-local striding: one block handles one Task
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    c[i] = a[i] + b[i];
  }
}

// Initialize jump table: null everything, install built-in op at slot 0
extern "C" __global__ void init_builtin_ops() {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < GPUOS_MAX_OPS) {
    g_op_table[idx] = nullptr;
    g_op_alias[idx] = idx; // identity mapping
  }
  if (idx == 0) {
    g_op_table[0] = op_add;
  }
}

// Persistent worker kernel: each thread acts as a consumer
extern "C" __global__ void persistent_worker(WorkQueue q) {
  if (q.capacity == 0) return;
  __shared__ Task s_task;
  __shared__ int s_has_work;
  while (atomicAdd(q.quit, 0) == 0) {
    if (threadIdx.x == 0) {
      int idx = atomicAdd(q.head, 1);
      int tail = atomicAdd(q.tail, 0);
      if (idx < tail) {
        s_task = q.tasks[idx % q.capacity];
        s_has_work = 1;
      } else {
        s_has_work = 0;
      }
    }
    __syncthreads();
    if (!s_has_work) {
      __nanosleep(1000);
      continue;
    }

    // Ensure we observe the latest function table contents
    __threadfence();
    OpFn fn = nullptr;
    if (s_task.op >= 0 && s_task.op < GPUOS_MAX_OPS) {
      int phys = atomicAdd(&g_op_alias[s_task.op], 0);
      if (phys >= 0 && phys < GPUOS_MAX_OPS) {
        unsigned long long p = atomicAdd((unsigned long long*)&g_op_table[phys], 0ULL);
        fn = (OpFn)p;
      }
    }
    if (fn) {
      fn(s_task);
      __syncthreads();
      if (threadIdx.x == 0) {
        atomicAdd(&g_processed_count, 1ULL);
      }
    }
    __syncthreads();
  }
}
